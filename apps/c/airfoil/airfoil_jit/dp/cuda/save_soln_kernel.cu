
#include <hip/hip_runtime.h>
//
// auto-generated by op2.py
//

//user function
__device__ void save_soln_gpu( const double *q, double *qold)
{
  for (int n = 0; n < 4; n++)
    qold[n] = q[n];

}

//C CUDA kernel function
__global__ void op_cuda_save_soln(
 const double* __restrict arg0,
 double* __restrict arg1,
 int set_size)
{
  //Process set elements
  for (int n = threadIdx.x+blockIdx.x*blockDim.x; n < set_size; n += blockDim.x*gridDim.x)
  {

    //user function call
    save_soln_gpu(arg0+n*4,
                  arg1+n*4
    );

  }
}

